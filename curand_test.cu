#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void generate_kernel(hiprandState *state, int M, unsigned int *randomInts)
{
    int blockId = blockIdx.x;
    if (threadIdx.x == 0) {
        hiprandState localState = state[blockId];
        unsigned int x = hiprand(&localState) % M;
        randomInts[blockId] = x;
        state[blockId] = localState;
    }
    __syncthreads();
    printf("Block %d, Thread %d: %u\n", blockId, threadIdx.x, randomInts[blockId]);
}

__global__ void setup_kernel(hiprandState *state, unsigned long long seed)
{
    int id = blockIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

int main(int argc, char *argv[])
{
    const unsigned int threadsPerBlock = 64;
    const unsigned int blockCount = 64;
    int M = 16777216; // Default value for M

    if (argc >= 2) {
        M = atoi(argv[1]);
    }

    hiprandState *devStates;
    unsigned int *devRandomInts;

    /* Allocate space for PRNG states and random integers on the device */
    CUDA_CALL(hipMalloc((void **)&devStates, blockCount * sizeof(hiprandState)));
    CUDA_CALL(hipMalloc((void **)&devRandomInts, blockCount * sizeof(unsigned int)));

    /* Setup PRNG states */
    setup_kernel<<<blockCount, 1>>>(devStates, time(NULL));

    /* Generate and print random numbers */
    generate_kernel<<<blockCount, threadsPerBlock>>>(devStates, M, devRandomInts);

    /* Cleanup */
    CUDA_CALL(hipFree(devStates));
    CUDA_CALL(hipFree(devRandomInts));
    CUDA_CALL(hipDeviceReset());

    return EXIT_SUCCESS;
}
